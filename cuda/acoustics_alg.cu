#include "hip/hip_runtime.h"
/*
 * Student:	Trascau Mihai
 * Grupa:	344C4
 * 
 * Lucrare:	Ecuatia undelor pentru acustica 2D
 * Fisier:	acoustics_alg.h
 * Descriere:	Fisier sursa care contine implementarile pentru algoritmul utilizat (in cazul nostru MDF pentru ecuatia propagarii undei)
 */ 

#include "acoustics.h"
#include <math.h>

extern "C" double **ua, **ub, **uc, gain;

double *xchg_gpu;

// Variabile globale pentru rularea unui scenariu
extern "C" int nx;
extern "C" int ny;
extern "C" double H;
extern "C" double MAX_TIME;
extern "C" double TIME_STEP;
extern "C" int SAVE_TIME;

extern "C" scenario_t scenario[MAX_SCENARIOS];
extern "C" int num_scenarios;
extern "C" int scn_index;

double P_0;
double tau;
double P_r[1000][1000];
double chi[200][200];
//__device__ double gpu_chi[1000][1000];
// TODO: take notes of this

// required by the export_to_vtk function
int in_structure(int x, int y)
{
	int i;
	for(i=0;i<scenario[scn_index].nr_struct;i++)
	{
		if(x > scenario[scn_index].structure[i].c_points[0][0] && x < scenario[scn_index].structure[i].c_points[3][0])
		if(y > scenario[scn_index].structure[i].c_points[0][1] && y < scenario[scn_index].structure[i].c_points[1][1])
			return 1;
	}
	return 0;
}

__global__ void free_path_loss_kernel(double amp, double *gpu_chi)
{
	printf("[GPU] Enter Free Path Loss Kernel\n");

	double G = 10;
	double sigma = 1;
	double L = 0.051; // Initial 1
	double f = 3 * 1000000;
	double lightspeed = 3 * 1000000;// missing one zero maybe
	double env_pent = 1.001;
	double lambda = lightspeed / f;
	double R;
	double P_r;
	//double gpu_chi[1000][1000];
	double tau = 1;
	double P_0 = tau * (amp * amp);	

	R = sqrt((double)(blockIdx.x * blockIdx.x + blockIdx.y * blockIdx.y));
	P_r = P_0 * G * G * sigma * lambda * lambda;
	P_r /= (pow(4 * M_PI, 3) * pow(R, env_pent) * L);
	gpu_chi[blockIdx.x * blockDim.x + blockIdx.y] = sqrt(P_r / P_0);
	gpu_chi[blockIdx.x * blockDim.x + blockIdx.y] /= 10;
	gpu_chi[blockIdx.x * blockDim.x + blockIdx.y] += 0.9;
	// TODO: check if this kind of addressing is ok. It doesn't seem to be
	printf("[GPU] gpu_chi[%d][%d]=%lf\n", blockIdx.x, blockIdx.y, gpu_chi[blockIdx.x * blockDim.x + blockIdx.y]);
//	gpu_chi[blockIdx.x][blockIdx.y] = sqrt(P_r / P_0);
//	gpu_chi[blockIdx.x][blockIdx.y] /= 10;
//	gpu_chi[blockIdx.x][blockIdx.y] += 0.9;
//	printf("[GPU] gpu_chi[%d][%d]=%lf\n", blockIdx.x, blockIdx.y, gpu_chi[blockIdx.x][blockIdx.y]);
	printf("[GPU] Exit Free Path Loss Kernel\n");
}

__device__ int is_source_gpu(int radius, int source_active,
		int src_x, int src_y)
{
	if (!source_active)
		return 0;
	if (sqrt(pow(src_x - blockIdx.x, 2) + pow(src_y - blockIdx.y, 2)) <= radius)
		return 1;
	return 0;
}

__global__ void set_all_zero_kernel(double *ua_gpu, double *ub_gpu, double *uc_gpu)
{
	ua_gpu[blockIdx.x * blockDim.x + blockIdx.y] = 0;
	ub_gpu[blockIdx.x * blockDim.x + blockIdx.y] = 0;
	uc_gpu[blockIdx.x * blockDim.x + blockIdx.y] = 0;
	// TODO: sync CPU after this -> move to utils.cu file
}

__global__ void wireless_src_pulse_kernel(int step, double amp,
	       	double MAX_TIME, double TIME_STEP,
		int radius, int source_active, int src_x, int src_y,
		double *ua_gpu, double *ub_gpu, double *uc_gpu)
{
	if (step < (int)(MAX_TIME / TIME_STEP) / 2){
		// Pulse source
		if (is_source_gpu(radius, 1, src_x, src_y))
			uc_gpu[blockIdx.x * blockDim.x + blockIdx.y] = amp * fabs(sin(step * M_PI/4));
	} else if (source_active){
		if (is_source_gpu(radius, source_active, src_x, src_y)) {
			ua_gpu[blockIdx.x * blockDim.x + blockIdx.y] = 0;
			ub_gpu[blockIdx.x * blockDim.x + blockIdx.y] = 0;
			uc_gpu[blockIdx.x * blockDim.x + blockIdx.y] = 0;
		}
		// All threads should reach this point before setting source_active -> need a thread barrier here. Or simply write 2 kernels and syncCPU. CPU should set source_active = 0 after freezing.
		source_active = 0;	
	}
}

__device__ int on_edge_gpu()
{
	if (0 == blockIdx.x && 0 != blockIdx.y && blockIdx.y != blockDim.x - 1)
		return N_EDGE;
	if (blockDim.x == blockIdx.y - 1 && blockIdx.y != 0 && blockIdx.y != blockDim.x - 1)
		return S_EDGE;
	if (0 == blockIdx.y && 0 != blockIdx.x && blockIdx.x != blockDim.y - 1)
		return W_EDGE;
	if (blockIdx.y == blockDim.x - 1 && blockIdx.x != 0 && blockIdx.x != blockDim.y - 1)
		return E_EDGE;
	return 0;
}

__device__ int on_corner_gpu()
{
	if (0 == blockIdx.x && blockIdx.y == 0)
		return NW_CORNER;
	if (0 == blockIdx.x && blockIdx.y == blockDim.x - 1)
		return NE_CORNER;
	if (blockDim.y - 1 == blockIdx.x && blockIdx.y == 0)
		return SW_CORNER;
	if (blockDim.y - 1 == blockIdx.x && blockIdx.y == blockDim.x - 1)
		return SE_CORNER;
	return 0;
}

__device__ int on_structure_edge_gpu()
{
	// TODO: this is tricky because we have structure
	// 		and it needs malloc
	return 0;
}

__device__ int in_structure_gpu()
{
	return 0;
}

__device__ int on_structure_corner_gpu()
{
	return 0;
}

__device__ double compute_edge_node_gpu(int side, double *ub_gpu)
{
	switch(side)
	{
		case N_EDGE:
			return ub_gpu[(blockIdx.x + 1) * blockDim.x + blockIdx.y];
		case E_EDGE:
			return ub_gpu[blockIdx.x * blockDim.x + (blockIdx.y - 1)];
		case S_EDGE:
			return ub_gpu[(blockIdx.x - 1) * blockDim.x + blockIdx.y];
		case W_EDGE:
			return ub_gpu[blockIdx.x * blockDim.x + (blockIdx.y + 1)];
		default:
			return 0;
	}
}

__device__ double compute_corner_node_gpu(int corner, double *ub_gpu)
{
	switch(corner)
	{
		case NW_CORNER:
			return (ub_gpu[blockIdx.x * blockDim.x + (blockIdx.y+1)] +
					ub_gpu[(blockIdx.x+1) * blockDim.x + blockIdx.y])/2;
		case NE_CORNER:
			return (ub_gpu[(blockIdx.x + 1) * blockDim.x + blockIdx.y] + 
					ub_gpu[blockIdx.x * blockDim.x + (blockIdx.y-1)])/2;
		case SE_CORNER:
			return (ub_gpu[blockIdx.x * blockDim.x + (blockIdx.y-1)] +
					ub_gpu[(blockIdx.x-1) * blockDim.x + blockIdx.y])/2;
		case SW_CORNER:
			return (ub_gpu[(blockIdx.x - 1) * blockDim.x + blockIdx.y] +
					ub_gpu[blockIdx.x * blockDim.x + (blockIdx.y + 1)])/2;
		default:
			return 0;
	}
}

__device__ double compute_structure_corner_node_gpu(int corner, double *ub_gpu)
{
	switch(corner)
	{
		case NW_CORNER:
			return (ub_gpu[blockIdx.x * blockDim.x + (blockIdx.y-1)] +
					ub_gpu[(blockIdx.x-1) * blockDim.x + blockIdx.y])/2;
		case NE_CORNER:
			return (ub_gpu[(blockIdx.x-1) * blockDim.x + blockIdx.y] +
					ub_gpu[blockIdx.x * blockDim.x + (blockIdx.y+1)])/2;
		case SE_CORNER:
			return (ub_gpu[blockIdx.x * blockDim.x + (blockIdx.y+1)] +
					ub_gpu[(blockIdx.x+1) * blockDim.x + blockIdx.y])/2;
		case SW_CORNER:
			return (ub_gpu[(blockIdx.x+1) * blockDim.x + blockIdx.y] +
					ub_gpu[blockIdx.x * blockDim.x + (blockIdx.y-1)])/2;
		default:
			return 0;
	}
}

__device__ double compute_structure_edge_node_gpu(int side, double *ub_gpu)
{
	switch(side)
	{
		case N_EDGE:
			return ub_gpu[(blockIdx.x-1) * blockDim.x + blockIdx.y];
		case E_EDGE:
			return ub_gpu[blockIdx.x * blockDim.x + (blockIdx.y+1)];
		case S_EDGE:
			return ub_gpu[(blockIdx.x+1) * blockDim.x + blockIdx.y];
		case W_EDGE:
			return ub_gpu[blockIdx.x * blockDim.x + (blockIdx.y-1)];
		default:
			return 0;
	}
}

__device__ double compute_node_gpu(double gain,
		double *ua_gpu, double *ub_gpu, double *uc_gpu)
{
	return (2 * ub_gpu[blockIdx.x * blockDim.x + blockIdx.y] -
			ua_gpu[blockIdx.x * blockDim.x + blockIdx.y] +
			gain * (ub_gpu[(blockIdx.x + 1) * blockDim.x + blockIdx.y] -
				4 * ub_gpu[blockIdx.x * blockDim.x + blockIdx.y] +
				ub_gpu[(blockIdx.x - 1) * blockDim.x + blockIdx.y] +
				ub_gpu[blockIdx.x * blockDim.x + (blockIdx.y + 1)] +
				ub_gpu[blockIdx.x * blockDim.x + (blockIdx.y - 1)]));
}

__global__ void wireless_propagate_kernel(double gain, int radius, int source_active,
	       	int src_x, int src_y,	
		double *ua_gpu, double *ub_gpu, double *uc_gpu, double *chi_gpu)
{
	int place;

	if (!on_corner_gpu() &&
		       !on_edge_gpu() &&
		       !is_source_gpu(radius, source_active, src_x, src_y) &&
		       !on_structure_edge_gpu() &&
		       !on_structure_corner_gpu() &&
		       !in_structure_gpu())
		uc_gpu[blockIdx.x * blockDim.x + blockIdx.y] = compute_node_gpu(
				gain, ua_gpu, ub_gpu, uc_gpu);
	else if (place = on_edge_gpu())
		uc_gpu[blockIdx.x * blockDim.x + blockIdx.y] = compute_edge_node_gpu(
				place, ub_gpu);
	else if (place = on_corner_gpu())
		uc_gpu[blockIdx.x * blockDim.x + blockIdx.y] = compute_corner_node_gpu(
				place, ub_gpu);
	else if (place = on_structure_edge_gpu())
		uc_gpu[blockIdx.x * blockDim.x + blockIdx.y] = compute_structure_edge_node_gpu(
				place, ub_gpu);
	else if (place = on_structure_corner_gpu())
		uc_gpu[blockIdx.x * blockDim.x + blockIdx.y] = compute_structure_corner_node_gpu(
				place, ub_gpu);

	ua_gpu[blockIdx.x * blockDim.x + blockIdx.y] = 0;

	uc_gpu[blockIdx.x * blockDim.x + blockIdx.y] *=
		chi_gpu[blockIdx.x * blockDim.x + blockIdx.y];
}


void free_path_loss()
{
	double G = 10;
	double sigma = 1;
	double L = 0.051; // Initial 1
	double f = 3 * 1000000;
	double lightspeed = 3 * 1000000;// missing one zero maybe
	double env_pent = 1.001;
	double lambda = lightspeed / f;
	double R;

	int x, y;
	for (x = 0; x < nx; x++){
		for (y = 0; y < ny; y++){

			R = sqrt(x * x + y * y);

			P_r[x][y] = P_0 * G * G * sigma * lambda * lambda;
			P_r[x][y] /= (pow(4 * M_PI, 3) * pow(R, env_pent) * L) ;
			chi[x][y] =  sqrt(P_r[x][y] / P_0);
			chi[x][y] /= 10;
			chi[x][y] += 0.9;
			//printf("Chi[%d][%d]=%lf\n", x, y, chi[x][y]);
		}
	}
}

void init_power(double amp)
{
	tau = 1;
	P_0 = tau * (amp * amp);
}

void s_compute_acoustics()
{
	printf("nx=%d ny=%d\n", nx, ny);
	double *gpu_chi;
	double *ua_gpu, *ub_gpu, *uc_gpu;
	hipError_t cuda_status;
	dim3 dimBlock(nx, ny);

	cuda_status = hipMalloc((void **)&gpu_chi, nx * ny * sizeof(double));
	if (hipSuccess != cuda_status){
		printf("Failed hipMalloc gpu_chi with message %s\n", hipGetErrorString(cuda_status));
	}
	cuda_status = hipMalloc((void **)&ua_gpu, nx * ny * sizeof(double *));
	if (hipSuccess != cuda_status){
		printf("Failed hipMalloc ua_gpu with message %s\n", hipGetErrorString(cuda_status));
	}
	cuda_status = hipMalloc((void **)&ub_gpu, nx * ny * sizeof(double *));
	if (hipSuccess != cuda_status){
		printf("Failed hipMalloc ub_gpu with message %s\n", hipGetErrorString(cuda_status));
	}
	cuda_status = hipMalloc((void **)&uc_gpu, nx * ny * sizeof(double *));
	if (hipSuccess != cuda_status){
		printf("Failed hipMalloc uc_gpu with message %s\n", hipGetErrorString(cuda_status));
	}

	set_all_zero_kernel<<<dimBlock, 1>>>(ua_gpu, ub_gpu, uc_gpu);
	cuda_status = hipPeekAtLastError();
	if (hipSuccess != cuda_status){
		printf("Failed launching set_all_zero_kernel  %s\n", hipGetErrorString(cuda_status));
	} else {
		printf("Managed to launch set_all_zero_kernel\n");
	}
	// I need to sync GPU with CPU here so that u* vectors are safely zeroized
	cuda_status = hipDeviceSynchronize();	
	free_path_loss_kernel<<<dimBlock, 1>>>(scenario[scn_index].source.p_amp, gpu_chi);
	cuda_status = hipPeekAtLastError();
	printf("[CPU] Launched Kernel - Blocking until GPU execution complete\n");
	cuda_status = hipDeviceSynchronize();

	printf("[CPU] Execued Kernel\n");
	cuda_status = hipMemcpy(chi, gpu_chi, 200 * 200 * sizeof(double), hipMemcpyDeviceToHost);
	if (hipSuccess != cuda_status){
		printf("Problem copying from cuda %s\n", hipGetErrorString(cuda_status));
	}
	// TODO: bug here
	// TODO: don't do useless copy here. Next kernel can reuse the gpu_chi var
	// Document about paraview

	int step = 0;
	int source_active = 1;
	int radius = scenario[scn_index].source.radius;
	
	/*
	for (i = 0; i < nx; i++)
		for (j = 0; j < ny; j++)
			printf("chi[%d][%d]=%lf\n", i, j, chi[i][j]);
	*/

	while(step < (int)(MAX_TIME/TIME_STEP))
	{
		// Pulse source
		wireless_src_pulse_kernel<<<dimBlock, 1>>>(
				step,
				scenario[scn_index].source.p_amp,
				MAX_TIME,
				TIME_STEP,
				radius,
				source_active,
				scenario[scn_index].source.x /* src_x */,
				scenario[scn_index].source.y /* src_y */,
				ua_gpu,
				ub_gpu,
				uc_gpu);
		if (step >= (int)(MAX_TIME / TIME_STEP) / 2)
			source_active = 0;
		hipDeviceSynchronize();
		
		// Propagate wave
		// TODO: include chi factor here
		wireless_propagate_kernel<<<dimBlock, 1>>>(
				gain,
				radius,
				source_active,
				scenario[scn_index].source.x,
				scenario[scn_index].source.y,
				ua_gpu,
				ub_gpu,
				uc_gpu,
				gpu_chi);
		hipDeviceSynchronize();

		// !!!!!! TODO 2: save time should be extremely rare here, maybe just once	
		/*if(step%SAVE_TIME == 0){
			hipMemcpy(ua, ua_gpu, nx * ny * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(ub, ub_gpu, nx * ny * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(uc, uc_gpu, nx * ny * sizeof(double), hipMemcpyDeviceToHost);
                        export_to_vtk(step);
			// Produces error coz ua is ** and ua_gpu is *
		}*/	
	
		xchg_gpu = ua_gpu;
		ua_gpu = ub_gpu;
		ub_gpu = uc_gpu;
		uc_gpu = xchg_gpu;
		
		step++;
	}
	
	hipFree(gpu_chi);
	hipFree(ua_gpu);
	hipFree(ub_gpu);
	hipFree(uc_gpu);
}
