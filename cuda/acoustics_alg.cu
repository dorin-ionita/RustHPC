#include "hip/hip_runtime.h"
/*
 * Student:	Trascau Mihai
 * Grupa:	344C4
 * 
 * Lucrare:	Ecuatia undelor pentru acustica 2D
 * Fisier:	acoustics_alg.h
 * Descriere:	Fisier sursa care contine implementarile pentru algoritmul utilizat (in cazul nostru MDF pentru ecuatia propagarii undei)
 */ 

#include "acoustics.h"
#include <math.h>

extern "C" double **ua, **ub, **uc, **xchg, gain;

// Variabile globale pentru rularea unui scenariu
extern "C" int nx;
extern "C" int ny;
extern "C" double H;
extern "C" double MAX_TIME;
extern "C" double TIME_STEP;
extern "C" int SAVE_TIME;

extern "C" scenario_t scenario[MAX_SCENARIOS];
extern "C" int num_scenarios;
extern "C" int scn_index;

int on_edge(int x, int y)
{
	if(x == 0 && y != 0 && y != nx-1)
		return N_EDGE;
	if(x == ny-1 && y != 0 && y != nx-1)
		return S_EDGE;
	if(y == 0 && x != 0 && x != ny-1)
		return W_EDGE;
	if(y == nx-1 && x != 0 && x != ny-1)
		return E_EDGE;
	return 0;
}

int on_corner(int x, int y)
{
	if(x == 0 && y == 0)
		return NW_CORNER;
	if(x == 0 && y == nx-1)
		return NE_CORNER;
	if(x == ny-1 && y == 0)
		return SW_CORNER;
	if(x == ny-1 && y == nx-1)
		return SE_CORNER;
	return 0;
}

int on_structure_edge(int x, int y)
{
	int i;
	for(i=0;i<scenario[scn_index].nr_struct;i++)
	{
		if(y > scenario[scn_index].structure[i].c_points[0][1] && y < scenario[scn_index].structure[i].c_points[1][1])
			if(x == scenario[scn_index].structure[i].c_points[0][0])
				return N_EDGE;
		if(x > scenario[scn_index].structure[i].c_points[1][0] && x < scenario[scn_index].structure[i].c_points[2][0])
			if(y == scenario[scn_index].structure[i].c_points[1][1])
				return E_EDGE;
		if(y > scenario[scn_index].structure[i].c_points[3][1] && y < scenario[scn_index].structure[i].c_points[2][1])
			if(x == scenario[scn_index].structure[i].c_points[3][0])
				return S_EDGE;
		if(x > scenario[scn_index].structure[i].c_points[0][0] && x < scenario[scn_index].structure[i].c_points[3][0])
			if(y == scenario[scn_index].structure[i].c_points[0][1])
				return W_EDGE;
	}
	return 0;
}

int on_structure_corner(int x, int y)
{
	int i;
	for(i=0;i<scenario[scn_index].nr_struct;i++)
	{
		if(x == scenario[scn_index].structure[i].c_points[0][0] && y == scenario[scn_index].structure[i].c_points[0][1])
			return NW_CORNER;
		if(x == scenario[scn_index].structure[i].c_points[1][0] && y == scenario[scn_index].structure[i].c_points[1][1])
			return NE_CORNER;
		if(x == scenario[scn_index].structure[i].c_points[2][0] && y == scenario[scn_index].structure[i].c_points[2][1])
			return SE_CORNER;
		if(x == scenario[scn_index].structure[i].c_points[3][0] && y == scenario[scn_index].structure[i].c_points[3][1])
			return SW_CORNER;
	}
	return 0;
}

int in_structure(int x, int y)
{
	int i;
	for(i=0;i<scenario[scn_index].nr_struct;i++)
	{
		if(x > scenario[scn_index].structure[i].c_points[0][0] && x < scenario[scn_index].structure[i].c_points[3][0])
		if(y > scenario[scn_index].structure[i].c_points[0][1] && y < scenario[scn_index].structure[i].c_points[1][1])
			return 1;
	}
	return 0;
}

double compute_node(int x, int y)
{
	return (2*ub[x][y] - ua[x][y] + gain * (ub[x+1][y] - 4*ub[x][y] + ub[x-1][y] + ub[x][y+1] + ub[x][y-1]));
}

double compute_edge_node(int i, int j, int side)
{
	switch(side)
	{
		case N_EDGE:
			return ub[i+1][j];
		case E_EDGE:
			return ub[i][j-1];
		case S_EDGE:
			return ub[i-1][j];
		case W_EDGE:
			return ub[i][j+1];
		default:
			return 0;
	}
}

double compute_corner_node(int i, int j, int corner)
{
	switch(corner)
	{
		case NW_CORNER:
			return (ub[i][j+1]+ub[i+1][j])/2;
		case NE_CORNER:
			return (ub[i+1][j]+ub[i][j-1])/2;
		case SE_CORNER:
			return (ub[i][j-1]+ub[i-1][j])/2;
		case SW_CORNER:
			return (ub[i-1][j]+ub[i][j+1])/2;
		default:
			return 0;
	}
}

double compute_structure_corner_node(int i, int j, int corner)
{
	switch(corner)
	{
		case NW_CORNER:
			return (ub[i][j-1]+ub[i-1][j])/2;
		case NE_CORNER:
			return (ub[i-1][j]+ub[i][j+1])/2;
		case SE_CORNER:
			return (ub[i][j+1]+ub[i+1][j])/2;
		case SW_CORNER:
			return (ub[i+1][j]+ub[i][j-1])/2;
		default:
			return 0;
	}
}

double compute_structure_edge_node(int i, int j, int side)
{
	switch(side)
	{
		case N_EDGE:
			return ub[i-1][j];
		case E_EDGE:
			return ub[i][j+1];
		case S_EDGE:
			return ub[i+1][j];
		case W_EDGE:
			return ub[i][j-1];
		default:
			return 0;
	}
}

int is_source(int x, int y, int radius, int source_active)
{
	if(!source_active)
		return 0;
	if(sqrt(pow(scenario[scn_index].source.x-x,2)+pow(scenario[scn_index].source.y-y,2)) <= radius)
		return 1;
	return 0;
}

void pulse_source(int radius, int step, double amp)
{
	int i,j;
	for(i=0;i<ny;i++)
	for(j=0;j<nx;j++)
		if(is_source(i,j,radius,1))
				uc[i][j] = amp*fabs(sin(step*M_PI/4));
}

double P_0;
double tau;
double P_r[1000][1000];
double chi[1000][1000];
//__device__ double gpu_chi[1000][1000];
// TODO: take notes of this

__global__ void free_path_loss_kernel(double amp, double *gpu_chi)
{
	printf("[GPU] Enter Free Path Loss Kernel\n");

	double G = 10;
	double sigma = 1;
	double L = 0.051; // Initial 1
	double f = 3 * 1000000;
	double lightspeed = 3 * 1000000;// missing one zero maybe
	double env_pent = 1.001;
	double lambda = lightspeed / f;
	double R;
	double P_r;
	//double gpu_chi[1000][1000];
	double tau = 1;
	double P_0 = tau * (amp * amp);	

	R = sqrt((double)(blockIdx.x * blockIdx.x + blockIdx.y * blockIdx.y));
	P_r = P_0 * G * G * sigma * lambda * lambda;
	P_r /= (pow(4 * M_PI, 3) * pow(R, env_pent) * L);
	gpu_chi[blockIdx.x * blockDim.x + blockIdx.y] = sqrt(P_r / P_0);
	gpu_chi[blockIdx.x * blockDim.x + blockIdx.y] /= 10;
	gpu_chi[blockIdx.x * blockDim.x + blockIdx.y] += 0.9;
	printf("[GPU] gpu_chi[%d][%d]=%lf\n", blockIdx.x, blockIdx.y, gpu_chi[blockIdx.x * blockDim.x + blockIdx.y]);
//	gpu_chi[blockIdx.x][blockIdx.y] = sqrt(P_r / P_0);
//	gpu_chi[blockIdx.x][blockIdx.y] /= 10;
//	gpu_chi[blockIdx.x][blockIdx.y] += 0.9;
//	printf("[GPU] gpu_chi[%d][%d]=%lf\n", blockIdx.x, blockIdx.y, gpu_chi[blockIdx.x][blockIdx.y]);
	printf("[GPU] Exit Free Path Loss Kernel\n");
}

void free_path_loss()
{
	double G = 10;
	double sigma = 1;
	double L = 0.051; // Initial 1
	double f = 3 * 1000000;
	double lightspeed = 3 * 1000000;// missing one zero maybe
	double env_pent = 1.001;
	double lambda = lightspeed / f;
	double R;

	int x, y;
	for (x = 0; x < nx; x++){
		for (y = 0; y < ny; y++){

			R = sqrt(x * x + y * y);

			P_r[x][y] = P_0 * G * G * sigma * lambda * lambda;
			P_r[x][y] /= (pow(4 * M_PI, 3) * pow(R, env_pent) * L) ;
			chi[x][y] =  sqrt(P_r[x][y] / P_0);
			chi[x][y] /= 10;
			chi[x][y] += 0.9;
			//printf("Chi[%d][%d]=%lf\n", x, y, chi[x][y]);
		}
	}
}

void init_power(double amp)
{
	tau = 1;
	P_0 = tau * (amp * amp);
}

void s_compute_acoustics()
{
	//init_power(scenario[scn_index].source.p_amp);
	double *gpu_chi;
	//double **chi = (double **)malloc(1000 * 1000 * sizeof(double));
	//free_path_loss();
	hipError_t cuda_status;
	hipMalloc((void **)&gpu_chi, 1000 * 1000 * sizeof(double));
	dim3 dimBlock(nx, ny);
	free_path_loss_kernel<<<dimBlock, 1>>>(scenario[scn_index].source.p_amp, gpu_chi);
	printf("[CPU] Launched Kernel - Blocking until GPU execution complete\n");
	hipDeviceSynchronize();

	printf("[CPU] Execued Kernel\n");
	cuda_status = hipMemcpy(chi, gpu_chi, 1000 * 1000 * sizeof(double), hipMemcpyDeviceToHost);
	if (hipSuccess != cuda_status){
		printf("Problem copying from cuda %s\n", hipGetErrorString(cuda_status));
	}
	// TODO: don't do useless copy here. Next kernel can reuse the gpu_chi var
	// Document about paraview

	int i,j;
	int step = 0;
	int source_active = 1;
	int place;
	int radius = scenario[scn_index].source.radius;
	
	for (i = 0; i < nx; i++)
		for (j = 0; j < ny; j++)
			printf("chi[%d][%d]=%lf\n", i, j, chi[i][j]);

	while(step < (int)(MAX_TIME/TIME_STEP))
	{
		if(step < (int)(MAX_TIME/TIME_STEP)/2)
			pulse_source(radius,step,scenario[scn_index].source.p_amp);
		else if(source_active)
		{
			for(i=0;i<ny;i++)
			for(j=0;j<nx;j++)
			{
				if(is_source(i,j,radius,source_active))
					uc[i][j] = ub[i][j] = ua[i][j] = 0;
			}
			source_active = 0;
		}
		// TODO 4: try to integrate this for in a single loop. Is it possible?
		
		for(i=0;i<ny;i++)
		for(j=0;j<nx;j++)
		{
			// TODO 2: Any pair (i,j) is indepenednt of all others (i, j) pairs
			// 		=> This means that each (i, j) can be a block 
			if(!on_corner(i,j) && !on_edge(i,j) && !is_source(i,j,radius,source_active) && !on_structure_edge(i,j) && !on_structure_corner(i,j) && !in_structure(i,j)){
				uc[i][j] = compute_node(i,j);
			}
			else if((place = on_edge(i,j)))
				uc[i][j] = compute_edge_node(i,j,place);
			else if((place = on_corner(i,j)))
				uc[i][j] = compute_corner_node(i,j,place);
			else if((place = on_structure_edge(i,j)))
				uc[i][j] = compute_structure_edge_node(i,j,place);
			else if((place = on_structure_corner(i,j)))
				uc[i][j] = compute_structure_corner_node(i,j,place);
			
			ua[i][j] = 0;
		}
	
		// TODO 2: save time should be extremely rare here, maybe just once	
		if(step%SAVE_TIME == 0)
                        export_to_vtk(step);	
		
		for (i = 0; i < nx; i++){
			for (j = 0; j < ny; j++){
				uc[i][j] *=  chi[i][j];
				// TODO 3: move this in the above for -> how about saving the data
			}
		}
		
		// TODO 2: sync here for cuda
		xchg = ua;
		ua = ub;
		ub = uc;
		uc = xchg;
		
		step++;
	}
	
	hipFree(gpu_chi);
}
